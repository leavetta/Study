#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void gInitializeStorage(float* storage_d){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int N = blockDim.x * gridDim.x;
    storage_d[i + j * N] = (float)(i + j * N);
}

__global__ void gTranspose0(float* storage_d, float* storage_d_t){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int N = blockDim.x * gridDim.x;
    storage_d_t[j + i * N] = storage_d[i + j * N];
}

void Output(float* a, int N){
     for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            fprintf(stdout, "%g\t", a[j + i * N]);
            fprintf(stdout, "\n");
    }
    fprintf(stdout,"\n\n\n");
}

int main() {
    int N = 256;
    int dim_of_threads = 32;
    int dim_of_blocks = N / dim_of_threads; 

    float *storage_d, *storage_d_t, *storage_h;

    hipMalloc((void**)&storage_d, N*N*sizeof(float));
    hipMalloc((void**)&storage_d_t, N*N*sizeof(float));
    storage_h=(float*)calloc(N*N, sizeof(float));

    gInitializeStorage <<< dim3(dim_of_blocks, dim_of_blocks), dim3(dim_of_threads, dim_of_threads) >>> (storage_d);
    
    hipDeviceSynchronize();
    memset(storage_h, 0.0, N * N * sizeof(float));
    hipMemcpy(storage_h, storage_d, N * N * sizeof(float), hipMemcpyDeviceToHost);
    Output(storage_h, N);

    gTranspose0 <<< dim3(dim_of_blocks, dim_of_blocks), dim3(dim_of_threads, dim_of_threads) >>> (storage_d, storage_d_t);
    hipDeviceSynchronize();
    memset(storage_h, 0.0, N * N * sizeof(float));
    hipMemcpy(storage_h, storage_d_t, N * N * sizeof(float), hipMemcpyDeviceToHost);
    Output(storage_h, N);

    hipFree(storage_d);
    hipFree(storage_d_t);
    free(storage_h);
        
    return 0;
}