#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_CHECK_RETURN(value) {\
    hipError_t _m_cudaStat = value;\
    if (_m_cudaStat != hipSuccess) {\
    fprintf(stderr, "Error %s at line %d in file %s\n",\
    hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
    exit(1);\
} }


__global__ void gTest1(float* a){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int I = gridDim.x * blockDim.x;
    //int J = gridDim.y * blockDim.y;
    a[i + j * I] = (float)(threadIdx.x + blockDim.y * blockIdx.x);
}

__global__ void gTest2(float* a){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    //int I = gridDim.x * blockDim.x;
    int J = gridDim.y * blockDim.y;
    a[j + i * J] = (float)(threadIdx.y + threadIdx.x * blockDim.y);
}


int main() {
    int n = 256;
    int threads_per_block = 32;
    int num_of_blocks = n / threads_per_block;

    float elapsedTime;
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float * a_gpu, * b_gpu, *result_a, *result_b;
    CUDA_CHECK_RETURN(hipMalloc((void**)&a_gpu, n * n * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void**)&b_gpu, n * n * sizeof(float)));
    result_a = (float*)calloc(n * n, sizeof(float));
    result_b = (float*)calloc(n * n, sizeof(float));

    hipEventRecord(start,0);

    gTest1 <<< dim3(num_of_blocks), dim3(threads_per_block) >>> (a_gpu);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    CUDA_CHECK_RETURN(hipGetLastError());
    hipEventElapsedTime(&elapsedTime,start,stop);
    fprintf(stderr,"gTest1 took %g \t\t num_of_blocks = %d \t\t threads_per_block = %d\n", elapsedTime, num_of_blocks, threads_per_block);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    CUDA_CHECK_RETURN(hipMemcpy(result_a, a_gpu, n * n * sizeof(float), hipMemcpyDeviceToHost));

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    gTest2 <<< dim3(num_of_blocks), dim3(threads_per_block) >>> (b_gpu);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    CUDA_CHECK_RETURN(hipGetLastError());
    hipEventElapsedTime(&elapsedTime,start,stop);
    fprintf(stderr,"gTest2 took %g \t\t num_of_blocks = %d \t\t threads_per_block = %d\n", elapsedTime, num_of_blocks, threads_per_block);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    CUDA_CHECK_RETURN(hipMemcpy(result_b, b_gpu, n * n * sizeof(float), hipMemcpyDeviceToHost));

    hipFree(a_gpu);
    hipFree(b_gpu);
    free(result_a);
    free(result_b);

    return 0;
}

