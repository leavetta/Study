#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_CHECK_RETURN(value) {\
    hipError_t _m_cudaStat = value;\
    if (_m_cudaStat != hipSuccess) {\
    fprintf(stderr, "Error %s at line %d in file %s\n",\
    hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
    exit(1);\
} }

/*__global__ void gInitializeStorage(float* storage_d){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int N = blockDim.x * gridDim.x;
    storage_d[i + j * N] = (float)(i + j * N);
}

__global__ void gTranspose0(float* storage_d, float* storage_d_t){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int N = blockDim.x * gridDim.x;
    storage_d_t[j + i * N] = storage_d[i + j * N];
}*/

__global__ void gTest1(float* a){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int I = gridDim.x * blockDim.x;
    //int J = gridDim.y * blockDim.y;
    a[i + j * I] = (float)(threadIdx.x + blockDim.y * blockIdx.x);
}

__global__ void gTest2(float* a){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    //int I = gridDim.x * blockDim.x;
    int J = gridDim.y * blockDim.y;
    a[j + i * J] = (float)(threadIdx.y + threadIdx.x * blockDim.y);
}

/*void Output(float* a, int N){
     for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            fprintf(stdout, "%g\t", a[j + i * N]);
            fprintf(stdout, "\n");
    }
    fprintf(stdout,"\n\n\n");
}*/

int main() {
    int n = 256;
    int threads_per_block = 32;
    //while(threads_per_block <= 1024){
        int num_of_blocks = n / threads_per_block; 
        
        float elapsedTime;
        hipEvent_t start,stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        float * a_gpu, * b_gpu, *result_a, *result_b;
        CUDA_CHECK_RETURN(hipMalloc((void**)&a_gpu, n * n * sizeof(float)));
        CUDA_CHECK_RETURN(hipMalloc((void**)&b_gpu, n * n * sizeof(float)));
        result_a = (float*)calloc(n * n, sizeof(float));
        result_b = (float*)calloc(n * n, sizeof(float));

        hipEventRecord(start,0);

        gTest1 <<< dim3(num_of_blocks), dim3(threads_per_block) >>> (a_gpu);
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        CUDA_CHECK_RETURN(hipGetLastError());
        hipEventElapsedTime(&elapsedTime,start,stop);
        fprintf(stderr,"gTest1 took %g \t\t num_of_blocks = %d \t\t threads_per_block = %d\n", elapsedTime, num_of_blocks, threads_per_block);
        hipEventDestroy(start);
        hipEventDestroy(stop);



        CUDA_CHECK_RETURN(hipMemcpy(result_a, a_gpu, n * n * sizeof(float), hipMemcpyDeviceToHost));

        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start,0);
        gTest2 <<< dim3(num_of_blocks), dim3(threads_per_block) >>> (b_gpu);
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        CUDA_CHECK_RETURN(hipGetLastError());
        hipEventElapsedTime(&elapsedTime,start,stop);
        fprintf(stderr,"gTest2 took %g \t\t num_of_blocks = %d \t\t threads_per_block = %d\n", elapsedTime, num_of_blocks, threads_per_block);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        CUDA_CHECK_RETURN(hipMemcpy(result_b, b_gpu, n * n * sizeof(float), hipMemcpyDeviceToHost));

        hipFree(a_gpu);
        hipFree(b_gpu);
        free(result_a);
        free(result_b);

       // threads_per_block *= 2;
    //}
    return 0;
}



     /*
     CUDA_CHECK_RETURN(hipMalloc((void**)&st_d, n * n * sizeof(float)));
        CUDA_CHECK_RETURN(hipMalloc((void**)&st_dt, n * n * sizeof(float)));
        storage_h = (float*)calloc(n * n, sizeof(float));
        storage_h = (float*)calloc(n * n, sizeof(float));
        storage_h = (float*)calloc(n * n, sizeof(float));
         = (float*)calloc(n * n, sizeof(float));
*/
  int n = 1000000;
    int threads_per_block = 128;
    //while(threads_per_block <= 1024){
        int num_of_blocks = n / threads_per_block;

        float elapsedTime;
        hipEvent_t start,stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        float * a_gpu, * b_gpu, *result_a, *result_b, *result_s;
        CUDA_CHECK_RETURN(hipMalloc((void**)&a_gpu, n * n * sizeof(float)));
        CUDA_CHECK_RETURN(hipMalloc((void**)&b_gpu, n * n * sizeof(float)));
        result_a = (float*)calloc(n * n, sizeof(float));
        result_b = (float*)calloc(n * n, sizeof(float));


__global__ void gInitializeStorage(float* storage_d){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int N = blockDim.x * gridDim.x;
    storage_d[i + j * N] = (float)(i + j * N);
}

__global__ void gTranspose0(float* storage_d, float* storage_d_t){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int N = blockDim.x * gridDim.x;
    storage_d_t[j + i * N] = storage_d[i + j * N];
}


void Output(float* a, int N){
     for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            fprintf(stdout, "%g\t", a[j + i * N]);
            fprintf(stdout, "\n");
    }
    fprintf(stdout,"\n\n\n");
}

