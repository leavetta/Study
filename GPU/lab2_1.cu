#include <cstdlib>
#include <iostream>
//#include <cuda_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_CHECK_RETURN(value) {\
    hipError_t _m_cudaStat = value;\
    if (_m_cudaStat != hipSuccess) {\
    fprintf(stderr, "Error %s at line %d in file %s\n",\
    hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
    exit(1);\
} }

__global__ void vecAdd_kernel(float * a, float * b, float * result, int n)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    a[i] = b[i] = i;
    if (i < n)
        result[i] = a[i] + b[i];
}

int main() {
    int n = 1000000;
    int threads_per_block = 8;
//    while(threads_per_block <= 1024){
        int num_of_blocks = n / threads_per_block;

        float elapsedTime;
        hipEvent_t start,stop; // встроенный тип данных – структура, для фиксации контрольных
    //точек
        hipEventCreate(&start); // инициализация
        hipEventCreate(&stop); // событий

        //float * a = new float[n],
        float * a_gpu, * b_gpu, *result_gpu;
        CUDA_CHECK_RETURN(hipMalloc((void**)&a_gpu, n * sizeof(float)));
        //float * b = new float[n], * b_gpu;
        CUDA_CHECK_RETURN(hipMalloc((void**)&b_gpu, n * sizeof(float)));
        float * result = new float[n];//, * result_gpu;
        CUDA_CHECK_RETURN(hipMalloc((void**)&result_gpu, n * sizeof(float)));
        //for (int i = 0; i < n; i++)
          //  a[i] = b[i] = i;
        //CUDA_CHECK_RETURN(cudaMemcpy(a_gpu, a, n * sizeof(float), cudaMemcpyHostToDevice));
        //CUDA_CHECK_RETURN(cudaMemcpy(b_gpu, b, n * sizeof(float), cudaMemcpyHostToDevice));
        hipEventRecord(start,0); // привязка события
    //const int block_size = 256;
    //int num_blocks = (n + block_size - 1) / block_size;
    //vecAdd_kernel <<< num_blocks, block_size >>> (a_gpu, b_gpu, result_gpu, n);
        vecAdd_kernel <<< dim3(num_of_blocks), dim3(threads_per_block) >>> (a_gpu, b_gpu, result_gpu, n);
        hipEventRecord(stop,0); // привязка события
        hipEventSynchronize(stop); // синхронизация по событию
    //cudaDeviceSynchronize();
        CUDA_CHECK_RETURN(hipGetLastError());
        hipEventElapsedTime(&elapsedTime,start,stop); // вычисление затраченного времени
        fprintf(stderr,"gTest took %g \t\t num_of_blocks = %d \t\t threads_per_block = %d\n", elapsedTime, num_of_blocks, threads_per_block);
        hipEventDestroy(start); // освобождение
        hipEventDestroy(stop); // памяти

        CUDA_CHECK_RETURN(hipMemcpy(result, result_gpu, n * sizeof(float), hipMemcpyDeviceToHost));
    //for (int i = 0; i < n; i ++)
        //printf("%g\n", result[i]);
        //delete [] a;
        //delete [] b;
        delete [] result;
        hipFree(a_gpu);
        hipFree(b_gpu);
        hipFree(result_gpu);
        threads_per_block *= 2;
//    }
    return 0;
}

     